#include "hip/hip_runtime.h"
#define IRASTER
#define IDRAW

//Pre include
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "settings.h"
#include "dnaStructures.cuh"
#include "cudaTools.cuh"

//Include dna helper functions
#include "dnaPoint.cuh"
#include "dnaTriangle.cuh"
#include "dnaColor.cuh"

//Include renderer
#include "rasterizer.cuh"

//Temporary for printing text
#include <stdio.h>

#include "ErrorCheck.h"

//Global variables
__device__ __constant__ Settings	g_settings;
__device__ __constant__ int*		g_triangleCounts;
__device__ __constant__ Triangle*	g_triangleData;
__device__ __constant__ Color*		g_colorData;
__device__ __constant__ uchar4*		g_bestBuffer;
__device__ __constant__ uchar4*		g_drawBuffer;
__device__ __constant__ int2*		g_rasterLines;
__device__ __constant__ uint2*		g_fitnessData;

texture<uchar4, 2, hipReadModeElementType> texTarget;
__device__ unsigned int g_Generation = 0;


__device__ inline unsigned int getIndex()
{
	//Island offset + strain offset
	return (blockDim.x * blockIdx.x) + threadIdx.x;
}

__device__ inline unsigned int indexToGenerationIndex(const unsigned int index)
{
	return (blockDim.x * gridDim.x * ((g_Generation)&1)) + index;
}

__device__ inline unsigned int indexToFutureIndex(const unsigned int index)
{
	return (blockDim.x * gridDim.x * ((g_Generation+1)&1)) + index;
}

__device__ inline unsigned int getGenerationStrainId()
{
	//Timeline offset + Island offset + strain offset
	return (blockDim.x * gridDim.x * (g_Generation&1)) + (blockDim.x * blockIdx.x) + threadIdx.x;
}

__device__ inline unsigned int getFutureStrainId()
{
	//Timeline+1 offset + Island offset + strain offset
	return (blockDim.x * gridDim.x * ((g_Generation+1)&1)) + (blockDim.x * blockIdx.x) + threadIdx.x;
}

__global__ void initProcess()
{
	const int strainId = getGenerationStrainId();

	//Grab global pointers
	int* triangleCounts = g_triangleCounts;
	Triangle* triangleData = g_triangleData;
	Color* colorData = g_colorData;

	//Set triangle count to 0 for strain
	unsigned int minTriangles = g_settings.mutationRanges.strainMinTriangles;
	triangleCounts[strainId] = minTriangles;

	//Initialize first X triangles and colors
	for (unsigned int i = 0; i < minTriangles; ++i)
	{
		unsigned int index = interleavedIndex(strainId, i, g_settings.mutationRanges.strainMaxTriangles);

		initTriangle(triangleData[index], g_settings);
		initColor(colorData[index], g_settings);
	}
}

__global__ void renderProcess()
{
	//Setup basics
	const int strainId = getGenerationStrainId();
	const int idx = getIndex();

	//Grab global pointers
	int* triangleCounts = g_triangleCounts;
	Triangle* triangleData = g_triangleData;
	Color* colorData = g_colorData;

	//Offset rasterbuffer
#ifndef IRASTER
	int2* rasterLines = &g_rasterLines[idx * g_settings.imageInfo.imageHeight];
#else
	int2* rasterLines = &g_rasterLines[idx];
#endif

	//Offset drawbuffer
	const int imagesize = g_settings.imageInfo.imageWidth * g_settings.imageInfo.imageHeight;
#ifndef IDRAW
	const int drawOffset = idx * imagesize;
	uchar4* drawBuffer = &g_drawBuffer[drawOffset];
#else
	uchar4* drawBuffer = &g_drawBuffer[idx];
#endif

	//Initialize raster
	int rasterStart = 0;
	int rasterEnd = g_settings.imageInfo.imageHeight-1;

	//Loop over triangles
	const int triangleCount = triangleCounts[strainId];
	for (int i = 0; i < triangleCount; ++i)
	{
		clearRaster(rasterLines, rasterStart, rasterEnd, g_settings);
		const unsigned int index = interleavedIndex(strainId, i, g_settings.mutationRanges.strainMaxTriangles);

		
		for (int j=0; j<3; ++j)
		{
			int2& point1 = triangleData[index].point[j];
			int2& point2 = triangleData[index].point[(j+1)%3];
			passLine(point1, point2, rasterLines, rasterStart, rasterEnd, g_settings);
		}
		//Set color
		const uchar4 color = colorData[index].components;
		const float alphascale = (float)color.w / 255.0f;

		float4 fcolor = make_float4(
			color.x * alphascale,
			color.y * alphascale,
			color.z * alphascale,
			0
		);

		//Render triangle
		renderRaster(rasterLines, rasterStart, rasterEnd, drawBuffer, fcolor, g_settings, strainId);
	}
}

__global__ void fitnessProcess()
{
	//Calculate fitness of strain
	const int strainId = getGenerationStrainId();
	const int idx = getIndex();
	const int strains = blockDim.x * gridDim.x;
	unsigned int fitness = 0;

	//Grab global pointers
	uint2* fitnessBuffer = g_fitnessData;

	//Drawbuffer
	const int height = g_settings.imageInfo.imageHeight;
	const int width = g_settings.imageInfo.imageWidth;

#ifndef IDRAW
	const int drawOffset = idx * width * height;
	const uchar4* drawBuffer = &g_drawBuffer[drawOffset];
#else
	const uchar4* drawBuffer = &g_drawBuffer[idx];
#endif

	int x = 0;
	int y = (height / blockDim.y) * threadIdx.y;
	const int ymax = (height / blockDim.y) * (threadIdx.y+1);
#ifndef IDRAW
	int index = y * width;
#else
	int index = strains * y * width;
#endif
	for (; y < ymax; ++y)
	{
		for (x = 0; x < width; ++x)
		{
			const uchar4 utarget = tex2D(texTarget, x, y);
			const uchar4 ustrain = drawBuffer[index];
			
			const int r = utarget.x - ustrain.x;
			const int g = utarget.y - ustrain.y;
			const int b = utarget.z - ustrain.z;

			fitness += (unsigned int)(r*r + g*g + b*b);
#ifndef IDRAW
			++index;
#else
			index += strains;
#endif
		}
	}
	
	fitnessBuffer[idx] = make_uint2(0, idx);
	__syncthreads();
	atomicAdd((unsigned int*)&fitnessBuffer[idx], fitness);
}

__global__ void evolveProcess()
{
	const int strainId = getGenerationStrainId();
	const int strainFutureId = getFutureStrainId();
	const int idx = getIndex();

	const int maxstrains = g_settings.generationInfo.strainCount * g_settings.generationInfo.islandCount;

	//Tournament selection
	//Pick a random strain from this island
	//(strainId + randomBetween(0, maxstrains-1)) % maxstrains;
	const int randomNumber = (idx + randomBetween(0, maxstrains-1)) % maxstrains;//(threadIdx.x + randomBetween(0, g_settings.generationInfo.strainCount-1)) % g_settings.generationInfo.strainCount;
	const int randomIdx = randomNumber;//(blockDim.x * blockIdx.x) + randomNumber; //scale id to be global
	const int randomGenerationId = indexToGenerationIndex(randomIdx);

	//Compare scores
	const int winnerId = (g_fitnessData[idx].x < g_fitnessData[randomIdx].x) ? strainId : randomGenerationId;
	const bool mutate = (winnerId != strainId);

	//Clone winning strain to future strain
	const int triangleCount = g_triangleCounts[winnerId];
	g_triangleCounts[strainFutureId] = triangleCount;
	for (int i = 0; i < triangleCount; ++i)
	{
		const unsigned int index = interleavedIndex(winnerId, i, g_settings.mutationRanges.strainMaxTriangles);
		const unsigned int indexFuture = interleavedIndex(strainFutureId, i, g_settings.mutationRanges.strainMaxTriangles);

		cloneTriangle(g_triangleData[indexFuture],	g_triangleData[index]);
		cloneColor(g_colorData[indexFuture],		g_colorData[index]);

		if(mutate)
		{
			//MUTATE POINTS
			for( int pointIdx = 0; pointIdx < 3; ++pointIdx)
			{
				int2& point = g_triangleData[indexFuture].point[pointIdx];
				if ( willMutate(g_settings.mutationRates.pointMinMoveMutationRate))
				{
					point.x = clamp(point.x + randomBetween(-g_settings.mutationRanges.pointMinMoveRange, g_settings.mutationRanges.pointMinMoveRange), 0, g_settings.imageInfo.imageWidth-1);
					point.y = clamp(point.y + randomBetween(-g_settings.mutationRanges.pointMinMoveRange, g_settings.mutationRanges.pointMinMoveRange), 0, g_settings.imageInfo.imageHeight-1);

				}

				if ( willMutate(g_settings.mutationRates.pointMidMoveMutationRate))
				{
					point.x = clamp(point.x + randomBetween(-g_settings.mutationRanges.pointMidMoveRange, g_settings.mutationRanges.pointMinMoveRange), 0, g_settings.imageInfo.imageWidth-1);
					point.y = clamp(point.y + randomBetween(-g_settings.mutationRanges.pointMidMoveRange, g_settings.mutationRanges.pointMinMoveRange), 0, g_settings.imageInfo.imageHeight-1);
				}

				if ( willMutate(g_settings.mutationRates.pointMaxMoveMutationRate))
				{
					point.x = fastrand() % g_settings.imageInfo.imageWidth;
					point.y = fastrand() % g_settings.imageInfo.imageHeight;
				}
			}

			//MUTATE COLORS
			uchar4& components = g_colorData[indexFuture].components;
			if ( willMutate(g_settings.mutationRates.redMutationRate))
			{
				components.x = randomBetween(g_settings.mutationRanges.redRangeMin, g_settings.mutationRanges.redRangeMax);
			}
			if ( willMutate(g_settings.mutationRates.greenMutationRate))
			{
				components.y = randomBetween(g_settings.mutationRanges.greenRangeMin, g_settings.mutationRanges.greenRangeMax);
			}
			if ( willMutate(g_settings.mutationRates.blueMutationRate))
			{
				components.z = randomBetween(g_settings.mutationRanges.blueRangeMin, g_settings.mutationRanges.blueRangeMax);
			}
			if ( willMutate(g_settings.mutationRates.alphaMutationRate))
			{
				components.w = randomBetween(g_settings.mutationRanges.alphaRangeMin, g_settings.mutationRanges.alphaRangeMax);
			}
		}
	}

	//Mutate if this strain was a loser
	if(mutate)
	{
		if (willMutate(g_settings.mutationRates.strainAddTriangleMutationRate))
		{
			int count = g_triangleCounts[strainFutureId];
			if (count < g_settings.mutationRanges.strainMaxTriangles)
			{
				const unsigned int indexFutureAdd = interleavedIndex(strainFutureId, count, g_settings.mutationRanges.strainMaxTriangles);

				initTriangle(g_triangleData[indexFutureAdd], g_settings);
				initColor(g_colorData[indexFutureAdd], g_settings);

				g_triangleCounts[strainFutureId]++;
			}
		}
		if ( willMutate(g_settings.mutationRates.strainRemoveTriangleMutationRate))
		{
			int count = g_triangleCounts[strainFutureId];
			if (count > g_settings.mutationRanges.strainMinTriangles)
			{
				//Choose random triangle to remove
				const unsigned int triangleIdx = fastrand() % count;

				//To remove a triangle, simply clone the last triangle in the list OVER the removed triangle
				const unsigned int indexFutureEnd = interleavedIndex(strainFutureId, count-1, g_settings.mutationRanges.strainMaxTriangles);
				const unsigned int indexFutureRemove = interleavedIndex(strainFutureId, triangleIdx, g_settings.mutationRanges.strainMaxTriangles);

				cloneTriangle(g_triangleData[indexFutureRemove], g_triangleData[indexFutureEnd]);
				cloneColor(g_colorData[indexFutureRemove], g_colorData[indexFutureEnd]);

				g_triangleCounts[strainFutureId]--;
			}
		}
	}
}

__global__ void initRNG()
{
	//No generations used, calculate idx directly
	int idx = getIndex();
	hiprand_init(1337, idx, 0, &g_randState[idx]);
}

extern "C" void	launch_cudaSetupRNG(dim3 grid, dim3 block)
{
	initRNG<<<grid, block>>>();
}

extern "C" void launch_cudaInitialize( dim3 grid, dim3 block )
{
	initProcess<<< grid, block >>>();
}

extern "C" void launch_cudaRender(dim3 grid, dim3 block)
{
	//ENGAGE!
	renderProcess<<< grid, block >>>();
}

extern "C" void launch_cudaFitness(dim3 grid, dim3 block, hipArray* targetArray )
{
	//Bind texture
	hipBindTextureToArray(texTarget, targetArray);
	struct hipChannelFormatDesc desc;
	hipGetChannelDesc(&desc, targetArray);

	//Run fitness function
	fitnessProcess<<< grid, block >>>();
}

__host__ void increaseGeneration()
{
	unsigned int gen;
	hipMemcpyFromSymbol(&gen, HIP_SYMBOL(g_Generation), sizeof(unsigned int), 0);
	gen++;
	hipMemcpyToSymbol(HIP_SYMBOL(g_Generation), &gen, sizeof(unsigned int), 0);
}

extern "C" void launch_cudaEvolve(dim3 grid, dim3 block )
{
	//Generate new strains
	evolveProcess<<< grid, block >>>();
	increaseGeneration();
}

struct uint2_sort_x
{
	__host__ __device__
	bool operator()(uint2 x, uint2 y)
	{
		return x.x < y.x;
	}
};

extern "C" uint2 getBestId(Settings &settings, void* fitnessData)
{
	//Sort fitness array
	int arraysize = settings.generationInfo.islandCount * settings.generationInfo.strainCount;

	try
	{
		thrust::device_ptr<uint2> dev_ptr((uint2*)fitnessData);
		thrust::sort(dev_ptr, dev_ptr+(arraysize-1), uint2_sort_x());
	}
	catch(thrust::system_error &e)
	{
		// output an error message and exit
		std::cerr  << e.what() << std::endl;
		exit(-1);
	}

	//Since the best Score as at the start of the block, copy first Uint2 to host and return it
	uint2 bestId;
	hipMemcpy(&bestId, (uint2*)fitnessData, sizeof(uint2), hipMemcpyDeviceToHost);

	return bestId;
}

extern "C" void uploadConstants(Settings& settings,
	hiprandState* randState,
	void* counts,
	void* triangles,
	void* colors,
	void* best,
	void* draw,
	void* raster,
	void* fitness
)
{
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_settings),		&settings,	sizeof(Settings)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_randState),		&randState, sizeof(hiprandState*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_triangleCounts),	&counts,	sizeof(int*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_triangleData),	&triangles, sizeof(Triangle*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_colorData),		&colors,	sizeof(Color*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_bestBuffer),		&best,		sizeof(uchar4*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_drawBuffer),		&draw,		sizeof(uchar4*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_rasterLines),		&raster,	sizeof(uint2*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_fitnessData),		&fitness,	sizeof(uint2*)));
}